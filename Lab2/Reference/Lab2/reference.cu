#include "hip/hip_runtime.h"
#include <wb.h>

#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));      \
      return -1;                                                          \
    }                                                                     \
  } while (0)

#define TILE_WIDTH 32

// Compute C = A * B
__global__ void matrixMultiply(float *A, float *B, float *C, 
                               int numARows, int numAColumns, 
                               int numBRows, int numBColumns, 
                               int numCRows, int numCColumns) {
  //@@ Insert code to implement matrix multiplication here

  /* Allocated shared memory matricies for subtile A and subtile B */
  __shared__ float subTile_A[TILE_WIDTH][TILE_WIDTH];
  __shared__ float subTile_B[TILE_WIDTH][TILE_WIDTH];

  /* Define thread identification info */
  int tx = threadIdx.x;
  int ty = threadIdx.y;

  /* Define global index */
  int global_col = blockIdx.x * blockDim.x + tx;
  int global_row = blockIdx.y * blockDim.y + ty;

  /* Declare and initialize an accumulator */
  float Pvalue = 0;
  
  /* Iterate thru tiles */
  for (int m = 0; m < ((TILE_WIDTH + numAColumns - 1) / TILE_WIDTH); m++) {
    /* Copy tile A from global to shared */
    if (global_row < numARows && (m * TILE_WIDTH + tx) < numAColumns) {
      subTile_A[ty][tx] = A[global_row * numAColumns + (m * TILE_WIDTH + tx)];
    } else {
      subTile_A[ty][tx] = 0.0;
    }
    /* Copy tile B from global to shared */
    if ((m * TILE_WIDTH + ty) < numBRows && global_col < numBColumns) {
      subTile_B[ty][tx] = B[(m * TILE_WIDTH + ty) * numBColumns + global_col];
    } else {
      subTile_B[ty][tx] = 0.0;
    }
    __syncthreads();
    
    /* Calculate Pvalue */
    for (int k = 0; k < TILE_WIDTH; k++) {
      Pvalue += subTile_A[ty][k] * subTile_B[k][tx];
    }
    __syncthreads();
  }
  /* Assert index is in bounds and write Pvalue back to global memory */
  if (global_row < numARows && global_col < numBColumns) {
    C[global_row * numCColumns + global_col] = Pvalue;
  }
  
}

int main(int argc, char **argv) {
  wbArg_t args;
  float *hostA; // The A matrix
  float *hostB; // The B matrix
  float *hostC; // The output C matrix
  float *deviceA;
  float *deviceB;
  float *deviceC;
  int numARows;    // number of rows in the matrix A
  int numAColumns; // number of columns in the matrix A
  int numBRows;    // number of rows in the matrix B
  int numBColumns; // number of columns in the matrix B
  int numCRows;    // number of rows in the matrix C (you have to set this)
  int numCColumns; // number of columns in the matrix C (you have to set
                   // this)

  args = wbArg_read(argc, argv);

  wbTime_start(Generic, "Importing data and creating memory on host");
  hostA = (float *)wbImport(wbArg_getInputFile(args, 0), &numARows,
                            &numAColumns);
  hostB = (float *)wbImport(wbArg_getInputFile(args, 1), &numBRows,
                            &numBColumns);
  //@@ Set numCRows and numCColumns
  numCRows = numARows;
  numCColumns = numBColumns;
    
  //@@ Allocate the hostC matrix
  hostC = (float*)malloc(numCRows * numCColumns * sizeof(float));

  wbTime_stop(Generic, "Importing data and creating memory on host");

  wbLog(TRACE, "The dimensions of A are ", numARows, " x ", numAColumns);
  wbLog(TRACE, "The dimensions of B are ", numBRows, " x ", numBColumns);

  wbTime_start(GPU, "Allocating GPU memory.");
  //@@ Allocate GPU memory here
  hipMalloc((void **)&deviceA, numARows * numAColumns * sizeof(float));
  hipMalloc((void **)&deviceB, numBRows * numBColumns * sizeof(float));
  hipMalloc((void **)&deviceC, numCRows * numCColumns * sizeof(float));

  wbTime_stop(GPU, "Allocating GPU memory.");

  wbTime_start(GPU, "Copying input memory to the GPU.");
  //@@ Copy memory to the GPU here
  hipMemcpy(deviceA, hostA, numARows * numAColumns * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(deviceB, hostB, numBRows * numBColumns * sizeof(float), hipMemcpyHostToDevice);
  
  wbTime_stop(GPU, "Copying input memory to the GPU.");

  //@@ Initialize the grid and block dimensions here
  dim3 dimBlock(TILE_WIDTH, TILE_WIDTH, 1);
  dim3 dimGrid(ceil((float)numCColumns / TILE_WIDTH), ceil((float)numCRows / TILE_WIDTH), 1);
  
  wbTime_start(Compute, "Performing CUDA computation");
  matrixMultiply<<<dimGrid, dimBlock>>>(deviceA, deviceB, deviceC, 
                                          numARows, numAColumns, 
                                          numBRows, numBColumns,
                                          numCRows, numCColumns);

  hipDeviceSynchronize();
  wbTime_stop(Compute, "Performing CUDA computation");

  wbTime_start(Copy, "Copying output memory to the CPU");
  //@@ Copy the GPU memory back to the CPU here
  hipMemcpy(hostC, deviceC, numCRows * numCColumns * sizeof(float), hipMemcpyDeviceToHost);
  
  wbTime_stop(Copy, "Copying output memory to the CPU");

  wbTime_start(GPU, "Freeing GPU Memory");
  //@@ Free the GPU memory here
  hipFree(deviceA);
  hipFree(deviceB);
  hipFree(deviceC);

  wbTime_stop(GPU, "Freeing GPU Memory");

  wbSolution(args, hostC, numCRows, numCColumns);

  free(hostA);
  free(hostB);
  free(hostC);

  return 0;
}
